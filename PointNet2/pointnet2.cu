//
// Created by hanm on 22-7-12.
//
#include "../device_common.cuh"
#include <iostream>
#include "ctime"
#include "fstream"
#include "vector"
#include "../host_common.h"
#include "../baseline_sampling.cuh"
#include "ball_query_gpu.cuh"
#include "group_gpu.cuh"
#include "mlp.cuh"

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("please run this program by the following parameter: sample_number filePath\n");
        return 0;
    }
    //check gpu

    check_GPU();

    int sample_number = atoi(argv[1]);
    std::string filename = argv[2];

    //read point
    std::ifstream fin(filename);
    if (!fin.is_open()) {
        std::cout << "file not exist" << std::endl;
        return 0;
    }
    std::vector<Point> point_data;
    int count = 0;
    if (fin.is_open()) {
        float xx, yy, zz;
        while (fin >> xx >> yy >> zz) {
            point_data.push_back({xx, yy, zz});
            count++;
        }
    }
    fin.close();
    const int point_data_size = point_data.size();

    hipEvent_t start, stop, fps_start, fps_end, group_end;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&fps_start);
    hipEventCreate(&fps_end);
    hipEventCreate(&group_end);
    int k = 20; //每个点的最多邻居数量
    int channelNum = 3;
    float radius = 200; // 1 x 200
    float (*coordinates) = new float[point_data_size * 3];
    float *d_coord;
    float *result;
    int * idx;
    float * group_out;
    float * mlp_out1;
    float * mlp_out2;
    float * mlp_out3;
    float (* mlp_result) = new float[(sample_number)*32*k];
    int cov_high = 1, cov_width = 1;

    for (int i = 0; i < point_data_size; i++) {
        coordinates[i * 3] = point_data[i].pos[0];
        coordinates[i * 3 + 1] = point_data[i].pos[1];
        coordinates[i * 3 + 2] = point_data[i].pos[2];
    }
    //warmup
    warmup<<<1, 1>>>();
    hipDeviceSynchronize();

    hipEventRecord(start);

    hipMalloc((void **) &d_coord, (point_data_size)*sizeof(float)*3);
    hipMalloc((void **) &result, (sample_number)*sizeof(float)*3);
    hipMalloc((void **) &idx, (sample_number)*sizeof(int)*k);

    hipMalloc((void **) &group_out, (sample_number)*sizeof(float)*3*k);

    hipMalloc((void **) &mlp_out1, (sample_number)*sizeof(float)*16*k);
    hipMalloc((void **) &mlp_out2, (sample_number)*sizeof(float)*16*k);
    hipMalloc((void **) &mlp_out3, (sample_number)*sizeof(float)*32*k);

    hipEventRecord(fps_start);
    hipMemcpy(d_coord,coordinates,point_data_size *sizeof(float )*3 ,hipMemcpyHostToDevice);
    //sample
    farthest_point_sampling(point_data_size,sample_number,d_coord,result);
    hipEventRecord(fps_end);
    //query
    ball_query(point_data_size, sample_number, radius, k, result, d_coord, idx);
    //group map
    group_points(channelNum, point_data_size, sample_number, k, d_coord, idx, group_out);
    hipEventRecord(group_end);
    //mlp
    mlp(group_out, mlp_out1, cov_high, cov_width, 3, 16, sample_number, k); // 3-> 16
    mlp(mlp_out1, mlp_out2, cov_high, cov_width, 16, 16, sample_number, k); // 16->16
    mlp(mlp_out2, mlp_out3, cov_high, cov_width, 16, 32, sample_number, k); // 16->32
    hipEventRecord(stop);

    hipMemcpy(mlp_result, mlp_out3, (sample_number)*sizeof(float)*32*k, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float milliseconds_fps,milliseconds_group,milliseconds_mlp;
    hipEventElapsedTime(&milliseconds_fps, fps_start, fps_end);
    hipEventElapsedTime(&milliseconds_group, fps_end, group_end);
    hipEventElapsedTime(&milliseconds_mlp, group_end, stop);

    hipError_t err;
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }

    std::cout << "Report:" << std::endl;
    std::cout << "    Type   :PointNet++(GPU)" << std::endl;
    std::cout << "    Points :" << point_data_size<< std::endl;
    std::cout << "    NPoint :" << sample_number << std::endl;
    std::cout << "    RunTime:" << milliseconds << "ms" << std::endl;
    std::cout << "       Sample: " << milliseconds_fps << "ms(" << (milliseconds_fps*100.0/milliseconds) << "%)" << std::endl;
    std::cout << "       Group: " << milliseconds_group << "ms(" << (milliseconds_group*100.0/milliseconds) << "%)" << std::endl;
    std::cout << "       MLP: " << milliseconds_mlp << "ms(" << (milliseconds_mlp*100.0/milliseconds) << "%)" << std::endl;
    std::cout << "    Param  :" << filename << std::endl;
    std::time_t time_result = std::time(NULL);
    std::cout << "  Timestamp:" << std::asctime(std::localtime(&time_result)) << std::endl;

    hipFree(d_coord);
    hipFree(result);
    hipFree(idx);
    hipFree(group_out);
    hipFree(mlp_out1);
    hipFree(mlp_out2);
    hipFree(mlp_out3);
    free(coordinates);
    free(mlp_result);
    return 0;
}