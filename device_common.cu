#include "hip/hip_runtime.h"
#include "device_common.cuh"

__global__ void warmup(){
    return;
}


void check_GPU(){
    hipError_t cudaStatus;
    int num = 0;
    cudaStatus = hipGetDeviceCount(&num);
    std::cout << "Number of GPU: " << num << std::endl;
    hipDeviceProp_t prop;
    if (num > 0) {
        hipGetDeviceProperties(&prop, 0);
        std::cout << "Device: " << prop.name << std::endl;
    }
}
