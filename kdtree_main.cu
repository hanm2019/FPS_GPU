#include "hip/hip_runtime.h"
//
// Created by hanm on 22-7-5.
//
#include <iostream>
#include "ctime"
#include "fstream"
#include "device_common.cuh"
#include "algorithm"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include "kdtree.cuh"

int main(int argc, char **argv) {
    if (argc != 4) {
        printf("please run this program by the following parameter: kdtree_high sample_number filePath\n");
        return 0;
    }

    //check gpu

    check_GPU();

    int kd_high = atoi(argv[1]);
    int sample_number = atoi(argv[2]);
    std::string filename =  argv[3];

    clock_t start_t, end_t;
    clock_t start_build_t, end_build_t;


    //read point
    std::ifstream fin(filename);
    if (!fin.is_open()) {
        std::cout << "file not exist" << std::endl;
        return 0;
    }
    thrust::host_vector<float3> point_data;
    int count = 0;
    if (fin.is_open()) {
        float xx, yy, zz;
        while (fin >> xx >> yy >> zz) {
            point_data.push_back({xx, yy, zz});
            count++;
        }
    }
    fin.close();
    const int point_data_size = point_data.size();

    //warmup
    warmup<<<1, 1>>>();
    hipDeviceSynchronize();

    //build

    start_build_t = clock();
    int bucketSize = 1 << kd_high;

    thrust::device_vector<float3> dPoints=point_data;
    float3 * ptr = thrust::raw_pointer_cast(&dPoints[0]);

    float3 * up;
    float3 * down;
    float3 * result;

    thrust::device_vector<float4> bucketTableVector(bucketSize);
    thrust::device_vector<int> bucketIndexVector(bucketSize);
    thrust::device_vector<int> bucketLengthVector(bucketSize);

    thrust::fill(bucketTableVector.begin(), bucketTableVector.end(), float4{0,0,0,1e20});
    thrust::fill(bucketIndexVector.begin(), bucketIndexVector.end(), 0);
    thrust::fill(bucketLengthVector.begin(), bucketLengthVector.end(), point_data_size);

    int * bucketIndex = thrust::raw_pointer_cast(&bucketIndexVector[0]);
    int * bucketLength = thrust::raw_pointer_cast(&bucketLengthVector[0]);
    float4 * bucketTable = thrust::raw_pointer_cast(&bucketTableVector[0]);

    hipMalloc((void **)&up, bucketSize*sizeof(float3));
    hipMalloc((void **)&down, bucketSize*sizeof(float3));
    hipMalloc((void **)&result, sample_number*sizeof(float3));


    buildKDTree(bucketIndex, bucketLength, ptr, kd_high, up, down);

    end_build_t = clock();
    //fps
    sample(bucketIndex, bucketLength, ptr, bucketSize, up, down, bucketTable, sample_number, result);
    end_t = clock();
    start_t = start_build_t;

    thrust::copy(dPoints.begin(), dPoints.end(), point_data.begin());

    //read point
    std::ofstream fout("kdtree.txt");
    if (!fout.is_open()) {
        std::cout << "file failed to open" << std::endl;
        return 0;
    }
    for(const auto& point : point_data){
        fout << point.x << " " << point.y << " " << point.z << std::endl;
    }

    fout.close();


    std::cout << "Report:" << std::endl;
    std::cout << "    Type   :kdline(GPU) high:" << kd_high << std::endl;
    std::cout << "    Points :" << point_data_size<< std::endl;
    std::cout << "    NPoint :" << sample_number << std::endl;
    std::cout << "    RunTime:" << (double) (end_t - start_t) << "us" << std::endl;
    std::cout << "    BuildTime:" << (double) (end_build_t - start_build_t) << "us" << std::endl;
    std::cout << "    Param  :" << filename << std::endl;
    std::time_t time_result = std::time(NULL);
    std::cout << "  Timestamp:" << std::asctime(std::localtime(&time_result)) << std::endl;

    hipFree(up);
    hipFree(down);
    hipFree(result);

    return 0;
}



////whichDim simply means which dimension we are sorting by, 0 = x, 1 = y, 2 = z
//int constructKD(thrust::device_vector<float3>& dPoints, int begin, int end,	compare_float3_x& comp_x, compare_float3_y& comp_y ,compare_float3_z& comp_z, int numLevels) {
//    int whichDim = 0;
//    thrust::detail::normal_iterator<thrust::device_ptr<float3>> maxx = thrust::max_element(dPoints.begin() + begin,
//                                                                                           dPoints.begin() + end,
//                                                                                           comp_x);
//    thrust::detail::normal_iterator<thrust::device_ptr<float3>> maxy = thrust::max_element(dPoints.begin() + begin,
//                                                                                           dPoints.begin() + end,
//                                                                                           comp_y);
//    thrust::detail::normal_iterator<thrust::device_ptr<float3>> maxz = thrust::max_element(dPoints.begin() + begin,
//                                                                                           dPoints.begin() + end,
//                                                                                           comp_z);
//    thrust::detail::normal_iterator<thrust::device_ptr<float3>> minx = thrust::min_element(dPoints.begin() + begin,
//                                                                                           dPoints.begin() + end,
//                                                                                           comp_x);
//    thrust::detail::normal_iterator<thrust::device_ptr<float3>> miny = thrust::min_element(dPoints.begin() + begin,
//                                                                                           dPoints.begin() + end,
//                                                                                           comp_y);
//    thrust::detail::normal_iterator<thrust::device_ptr<float3>> minz = thrust::min_element(dPoints.begin() + begin,
//                                                                                           dPoints.begin() + end,
//                                                                                           comp_z);
//
//    float rangeX = static_cast<float3>(*maxx).x - static_cast<float3>(*minx).x;
//    float rangeY = static_cast<float3>(*maxy).y - static_cast<float3>(*miny).y;
//    float rangeZ = static_cast<float3>(*maxz).z - static_cast<float3>(*minz).z;
//
//    if (rangeX > rangeY && rangeX > rangeZ) {
//        whichDim = 0;
//    } else {
//        if (rangeY > rangeX && rangeY > rangeZ) {
//            whichDim = 1;
//        } else {
//            if (rangeZ > rangeX && rangeZ > rangeY) {
//                whichDim = 2;
//            } else {
//                whichDim = 0;
//            }
//        }
//    }
//    switch(whichDim)
//    {
//        case 0:
//            thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_x);
//            break;
//        case 1:
//            thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_y);
//            break;
//        case 2:
//            thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_z);
//            break;
//        default:
//            printf("You shouldn't be here; i.e. wrong case number");
//            break;
//    }
//
//    numLevels--;
//    int numOfPoints = end-begin;
//    int lowerBound = ((int)numOfPoints/2)+begin;
//    int upperBound = ((int)numOfPoints/2)+1+begin;
//    int toReturn=0;
//    if(numLevels>0)
//    {
//        toReturn=constructKD(dPoints, begin, lowerBound, comp_x, comp_y, comp_z, numLevels);
//        toReturn=constructKD(dPoints, upperBound, end, comp_x, comp_y, comp_z, numLevels);
//    }
//    return toReturn;
//
//}
//int constructKD(thrust::device_vector<float3>& dPoints, int whichDim, int begin, int end,	compare_float3_x& comp_x, compare_float3_y& comp_y ,compare_float3_z& comp_z, int numLevels) {
//
//    switch(whichDim)
//    {
//        case 0:
//            thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_x);
//            break;
//        case 1:
//            thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_y);
//            break;
//        case 2:
//            thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_z);
//            break;
//        default:
//            printf("You shouldn't be here; i.e. wrong case number");
//            break;
//    }
//
//    numLevels--;
//    int numOfPoints = end-begin;
//    int lowerBound = ((int)numOfPoints/2)+begin;
//    int upperBound = ((int)numOfPoints/2)+1+begin;
//    int toReturn=0;
//    if(numLevels>0)
//    {
//        toReturn=constructKD(dPoints, (whichDim + 1) % 3 , begin, lowerBound, comp_x, comp_y, comp_z, numLevels);
//        toReturn=constructKD(dPoints, (whichDim + 1) % 3 ,upperBound, end, comp_x, comp_y, comp_z, numLevels);
//    }
//    return toReturn;
//}